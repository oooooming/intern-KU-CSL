#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C++" {
#include "glife.h"
}
#include <hip/hip_runtime.h>

// HINT: YOU CAN USE THIS METHOD FOR ERROR CHECKING
// Print error message on CUDA API or kernel launch
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                    msg, hipGetErrorString(__err), \
                    __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
        } \
    } while (0)

// TODO: YOU MAY NEED TO USE IT OR CREATE MORE
__device__ int getNeighbors(int* grid, int tot_rows, int tot_cols,
        int row, int col) {
    int numOfNeighbors = 0;
    int dx[] = {-1, -1, -1, 0, 0, 1, 1, 1};
    int dy[] = {-1, 0, 1, -1, 1, -1, 0, 1};
    
    for (int i = 0; i < 8; i++) {
        int newRow = row + dx[i];
        int newCol = col + dy[i];
        
        if (newRow >= 0 && newRow < tot_rows && newCol >= 0 && newCol < tot_cols) {
            numOfNeighbors += grid[newRow * tot_cols + newCol]; // 1D for CUDA
        }
    }
    
    return numOfNeighbors;
}

// TODO: YOU NEED TO IMPLEMENT KERNEL TO RUN ON GPU DEVICE 
__global__ void kernel()
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y & blockDim.y + threadIdx.y;
    
    if (idx < tot_rows && idy < tot_cols) {
      int neighbors = getNeighbors(grid, tot_rows, tot_cols, idx, idy);
      
      // Game of Life rules
      if (grid[idx * tot_cols + idy] == 1) {
          if (neighbors == 2 || neighbors == 3)
              newGrid[idx * tot_cols + idy] = 1;
          else 
              newGrid[idx * tot_cols + idy] =0;
      } else {
          if (neighbors == 3)
              newGrid[idx * tot_cols + idy] = 1;
          else
              newGrid[idx * tot_cols + idy] = 0;
      }
    }
}

void cuda_dump(int *grid, int tot_rows, int tot_cols)
{
    printf("===============================\n");
    for (int i = 0; i < tot_rows; i++) {
        printf("[%d] ", i);
        for (int j = 0; j < tot_cols; j++) {
            if (grid[i * tot_cols + j])
                printf("*");
            else
                printf("o");
        }
        printf("\n");
    }
    printf("===============================\n");
}

void cuda_dump_index(int *grid, int tot_rows, int tot_cols)
{
    printf(":: Dump Row Column indices\n");
    for (int i = 0; i < tot_rows; i++) {
        for (int j = 0; j < tot_cols; j++) {
            if (grid[i * tot_cols + j])
                printf("%d %d\n", i, j);
        }
    }
}

// TODO: YOU NEED TO IMPLEMENT ON CUDA VERSION
uint64_t runCUDA(int rows, int cols, int gen, 
                 GameOfLifeGrid* g_GameOfLifeGrid, int display)
{
    hipSetDevice(0); // DO NOT CHANGE THIS LINE 

    uint64_t difft;

    // ---------- TODO: CALL CUDA API HERE ----------
    
    // allocate GPU memory
    int* d_grid;
    int* d_newGrid;
    hipMalloc(&d_grid, rows * cols * sizeof(int));
    hipMalloc(&d_newGrid, rows * cols * sizeof(int));
    
    // transfer data from CPU to GPU
    hipMemcpy(d_grid, g_GameOfLifeGrid->getGrid(), rows * cols * sizeof(int), hipMemcpyHostToDevice);

    // Start timer for CUDA kernel execution
    difft = dtime_usec(0);
    // ----------  TODO: CALL KERNEL HERE  ----------

    // run CUDA kernel
    dim3 blockSize(16, 16);
    dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, (rows + blockSize.y - 1) / blockSize.y);

    for (int i = 0; i < gen; i++) {
        kernel<<<gridSize, blockSize>>>(d_grid, d_newGrid, rows, cols);
        cudaCheckErrors("CUDA kernel launch failed");

        // update new state GPU -> GPU
        hipMemcpy(d_grid, d_newGrid, rows * cols * sizeof(int), hipMemcpyDeviceToDevice);
    }

    // Finish timer for CUDA kernel execution
    difft = dtime_usec(difft);

    // transfer data GPU to CPU
    hipMemcpy(g_GameOfLifeGrid->getGrid(), d_grid, rows * cols * sizeof(int), hipMemcpyDeviceToHost);

    // deallocate GPU memory
    hipFree(d_grid);
    hipFree(d_newGrid);

    // Print the results
    if (display) {
        cuda_dump();
        cuda_dump_index();
    }
    return difft;
}
#endif
